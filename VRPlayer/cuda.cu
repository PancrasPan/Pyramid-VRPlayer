#include "hip/hip_runtime.h"
#include <opencv2/core/cuda_types.hpp>
#include <opencv2/cudev/common.hpp>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <hip/hip_vector_types.h>
using namespace cv;
using namespace cv::cuda;
//自定义内核函数

__global__ void combine_kernel(const PtrStepSz<uchar4> imageL,
    const PtrStepSz<uchar4> imageR,
    const PtrStepSz<float> flowMagL,
    const PtrStepSz<float> flowMagR,
    PtrStep<uchar4> dst){
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if(x < imageL.cols && y < imageL.rows)
    {
        uchar4 colorL = imageL(y,x);
        uchar4 colorR = imageR(y,x);
        unsigned char outAlpha;
        if(colorL.w > colorR.w)
        {
            if(colorL.w / 255.0f > 0.1)
                outAlpha = 255;
            else
                outAlpha = 0;
        }
        else
        {
            if(colorR.w / 255.0f > 0.1)
                outAlpha = 255;
            else
                outAlpha = 0;
        }
        uchar4 colorMixed;
        if (colorL.w == 0 && colorR.w == 0) {
            colorMixed = make_uchar4(0, 0, 0, outAlpha);
        } else if (colorL.w == 0) {
            colorMixed = make_uchar4(colorR.x, colorR.y, colorR.z, outAlpha);
        } else if (colorR.w == 0) {
            colorMixed = make_uchar4(colorL.x, colorL.y, colorL.z, outAlpha);
        } else {
            const float magL = flowMagL(y,x) / float(imageL.cols);
            const float magR = flowMagR(y,x) / float(imageL.cols);
            float blendL = float(colorL.w);
            float blendR = float(colorR.w);
            float norm = blendL + blendR;
            blendL /= norm;
            blendR /= norm;
            const float colorDiff =
              (abs(colorL.x - colorR.x) +
               abs(colorL.y - colorR.y) +
               abs(colorL.z - colorR.z)) / 255.0f;
            const float kColorDiffCoef = 10.0f;
            const float kSoftmaxSharpness = 10.0f;
            const float kFlowMagCoef = 20.0f; // NOTE: this is scaled differently than the test version due to normalizing magL & magR by imageL.cols
            const float deghostCoef = tanhf(colorDiff * kColorDiffCoef);
            const double expL = exp(kSoftmaxSharpness * blendL * (1.0 + kFlowMagCoef * magL));
            const double expR = exp(kSoftmaxSharpness * blendR * (1.0 + kFlowMagCoef * magR));
            const double sumExp = expL + expR + 0.00001;
            const float softmaxL = float(expL / sumExp);
            const float softmaxR = float(expR / sumExp);
            colorMixed = make_uchar4(
                float(colorL.x)* (blendL * (1-deghostCoef) + softmaxL * deghostCoef) + float(colorR.x)*(blendR * (1-deghostCoef) + softmaxR * deghostCoef),
                float(colorL.y)* (blendL * (1-deghostCoef) + softmaxL * deghostCoef) + float(colorR.y)*(blendR * (1-deghostCoef) + softmaxR * deghostCoef),
                float(colorL.z)* (blendL * (1-deghostCoef) + softmaxL * deghostCoef) + float(colorR.z)*(blendR * (1-deghostCoef) + softmaxR * deghostCoef),  
                255);         
        }
        dst(y, x) = colorMixed;
        // uchar4 v = imageL(y,x);
        // dst(y,x) = make_uchar4(v.x,v.y,v.z,255);
    }
}

void combine_caller(const PtrStepSz<uchar4>& imageL,
    const PtrStepSz<uchar4>& imageR,
    const PtrStepSz<float>& flowMagL,
    const PtrStepSz<float>& flowMagR,
    PtrStep<uchar4> dst,hipStream_t stream){
    dim3 block(32,8);
    dim3 grid((imageL.cols + block.x - 1)/block.x,(imageL.rows + block.y - 1)/block.y);

    combine_kernel<<<grid,block,0,stream>>>(imageL,imageR,flowMagL,flowMagR, dst);
    if(stream == 0)
        hipDeviceSynchronize();
}

__global__ void shift_kernel(const PtrStepSz<float> shiftMat,
    PtrStep<uchar4> dst){
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if(x < shiftMat.cols && y < shiftMat.rows)
    {
        uchar4 image = dst(y,x);
        float alpha = shiftMat(y,x);
        image.w = (int)(image.w * alpha);
        dst(y,x) = image;
    }
}


void alpha_cuda_caller(const PtrStepSz<float>& shiftMat,
    PtrStep<uchar4> dst,
    hipStream_t stream){
    dim3 block(32,8);
    dim3 grid((shiftMat.cols + block.x - 1)/block.x,(shiftMat.rows + block.y - 1)/block.y);

    shift_kernel<<<grid,block,0,stream>>>(shiftMat,dst);
    if(stream == 0)
        hipDeviceSynchronize();
}
